/**
 *   CS6023: GPU Programming
 *   Assignment 2
 *
 *   Please don't change any existing code in this file.
 *
 *   Please add necessary memory APIs for your implementation. Use cudaFree()
 *   to free up memory as soon as you're done with an allocation.
 *   This will ensure that you don't run out of memory while running
 *   large test cases. Use the minimum required memory for your
 *   implementation. DO NOT change the kernel configuration parameters.
 */

#include <chrono>
#include <fstream>
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>

using namespace std;

using std::cin;
using std::cout;

typedef long long ll;

__constant__ int filter[2401];

__global__ void convolute(int *globalMatrix, int totalNums, int offset, int perThread, int threadsReq, long int *ans, int m, int n, int k)
{
    __shared__ int matrix[6000];
    unsigned int t_id = blockIdx.x * blockDim.x + threadIdx.x;
    // if (blockIdx.x != 2)
    //     return;
    // printf("%d ", t_id);
    if (t_id < 1024 * blockIdx.x + threadsReq)
    {
        int loc = 1024 * blockIdx.x + (t_id - 1024 * blockIdx.x) * (perThread);
        // printf("%d ",(1024*blockIdx.x));
        // printf("%d %d %d\n",t_id,loc,perThread);

        int temp = perThread;
        while (temp--)
        {
            if (loc >= 0 && loc < m * n)
            {
                // printf("%d %d\n", t_id, loc);
                matrix[loc - 1024 * blockIdx.x] = globalMatrix[loc];
            }
            loc++;
        }
    }

    __syncthreads();
    // if (t_id != 3059)
    // {
    //     /* code */
    //     return;
    // }

    // if(t_id==2048){
    //     for(int i=0;i<6000;i++)
    //         printf("%d ",matrix[i]);
    //     printf("\n");
    // }
    // return;

    int g_Row = t_id / n;
    int g_Col = t_id % n;
    int start_R = g_Row - k / 2;
    int start_C = g_Col - k / 2;

    int sum = 0;
    for (int i = 0; i < k; i++)
    {
        for (int j = 0; j < k; j++)
        {
            int f_ind = i * k + j;
            int r = start_R + i;
            int c = start_C + j;
            int m_ind = r * n + c;
            int val;
            if (r < 0 || r >= m || c < 0 || c >= n)
                val = 0;
            else
            {
                int s_id = m_ind - 1024 * blockIdx.x;
                // if (s_id >= blockIdx.x * 1024 && s_id < blockIdx.x * 1024 + totalNums)
                if (s_id >= 0 && s_id < offset){
                    // printf("%d ",s_id);
                    val = matrix[s_id];
                }
                else
                    val = globalMatrix[m_ind];
                sum += (val * filter[f_ind]);
                // printf("%d %d %d %d %d %d\n", s_id, m_ind, filter[f_ind], globalMatrix[m_ind], val, sum);
                // printf("%d %d %d %d %d %d %d %d %d\n", i, j, s_id, matrix[s_id], m_ind, sum, val, f_ind, filter[f_ind]);
                // if (t_id==3059)
                // {
                //     /* code */
                //     for(int i=0;i<2041;i++)
                //     printf("%d ",filter[i]);
                // }

                // printf("%d %d\n",i,j);
            }
        }
    }
    // printf("%d %d\n", g_Row, g_Col);
    // printf("%d %d \n", t_id, sum);
    __syncthreads();
    if (g_Row * n + g_Col < m * n)
        ans[g_Row * n + g_Col] = sum;
}

int main(int argc, char **argv)
{

    int m, n, k;
    cin >> m >> n >> k;

    long int *h_mat = new long int[m * n];
    long int *h_filter = new long int[k * k];

    long int *h_ans = new long int[m * n];

    for (long int i = 0; i < m * n; i++)
    {
        cin >> h_mat[i];
    }

    for (long int i = 0; i < k * k; i++)
    {
        cin >> h_filter[i];
    }

    /**
     *
     * DO NOT CHANGE ANYTHING ABOVE THIS LINE
     *
     **/

    /****************************************************Start Here***********************************************************/
    int *matrix;
    long int *ans;

    hipMemcpyToSymbol(HIP_SYMBOL(filter), h_filter, k * k * sizeof(long int), 0, hipMemcpyHostToDevice);

    hipMalloc(&matrix, m * n * sizeof(long int));
    hipMemcpy(matrix, h_mat, m * n * sizeof(long int), hipMemcpyHostToDevice);
    hipMalloc(&ans, m * n * sizeof(long int));

    int blocksReq = ceil((m * n) / 1024.0);
    int computableRows = ceil(1024.0 / n);
    computableRows = computableRows >= m ? m : computableRows;
    int rowsToBestored = computableRows;

    if (computableRows != m && (k / 2) * n < 6 * 1024)
        rowsToBestored += k / 2;
    else if (computableRows != m)
        rowsToBestored += (5 * 1024) / n;

    int size = rowsToBestored * n;
    int offset = rowsToBestored * n;
    int perThread = ceil(size / 1024.0);
    int threadsReq = size / perThread;
    // printf("%d %d %d %d %d %d %d", blocksReq, computableRows, rowsToBestored, size, offset, perThread, threadsReq);
    auto start = std::chrono::high_resolution_clock::now(); // keep it just before the kernel launch

    // printf("%d \n",size);
    convolute<<<blocksReq, 1024>>>(matrix, size, offset, perThread, threadsReq, ans, m, n, k);

    hipMemcpy(h_ans, ans, m * n * sizeof(long int), hipMemcpyDeviceToHost);

    auto end = std::chrono::high_resolution_clock::now(); // keep it just after the kernel launch

    /*$$$$$$$$$$$$$$$$$$$$$$$$Make sure your final output from the device is stored in h_ans.$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$*/
    std::chrono::duration<double> elapsed1 = end - start;
    /**
     *
     * DO NOT CHANGE ANYTHING BELOW THIS LINE
     *
     */

    std::ofstream file("cuda.out");
    if (file.is_open())
    {
        for (long int i = 0; i < m; i++)
        {
            for (long int j = 0; j < n; j++)
            {
                file << h_ans[i * n + j] << " ";
            }
            file << "\n";
        }
        file.close();
    }
    else
    {
        std::cout << "Unable to open file";
    }

    std::ofstream file2("cuda_timing.out");
    if (file2.is_open())
    {
        file2 << elapsed1.count() << "\n";
        file2.close();
    }
    else
    {
        std::cout << "Unable to open file";
    }

    return 0;
}
